#include "hip/hip_runtime.h"
/*
 * Cuda kernels for quantization and mixed-precision packing
 */

#include <torch/extension.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


using torch::IntArrayRef;
using torch::Tensor;
using torch::autograd::tensor_list;

/****************************************/
/****** Pack/Unpack Mixed Precision *****/
/****************************************/
template <typename scalar_t>
__global__ void compute_scale_mixed_precision_kernel(const int32_t* __restrict__ bits,
                                                     const scalar_t* __restrict__ min,
                                                     const scalar_t* __restrict__ max,
                                                     scalar_t* __restrict__ scale,
                                                     int N,
                                                     int num_groups) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < N * num_groups) {
    scale[id] = ((scalar_t)((1 << bits[id / num_groups]) - 1)) / (max[id] - min[id] + 2e-6);
  }
}


template <typename scalar_t>
__global__ void pack_mixed_precision_kernel(const int32_t* __restrict__ bits,
                                            const int32_t* __restrict__ prefix_sum,
                                            const scalar_t* __restrict__ data,
                                            const scalar_t* __restrict__ scale,
                                            const scalar_t* __restrict__ min,
                                            int32_t* __restrict__ packed,
                                            std::pair<uint64_t, uint64_t> seeds,
                                            int N,
                                            int num_groups,
                                            int group_size) {
  extern __shared__ int packed_shared[];

  const int n = blockIdx.y;
  const int group_id = blockIdx.x;
  const int d = threadIdx.x;
  const int id = (n * num_groups + group_id) * group_size + d;
  const int shared_len = group_size * bits[n] / (sizeof(int32_t) * 8);

  if (threadIdx.x * 2 < shared_len) {
    reinterpret_cast<int2*>(packed_shared)[threadIdx.x] = make_int2(0, 0);
  }

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, id, seeds.second, &state);
  const float noise = hiprand_uniform(&state);

  const int val = __float2int_rn(fmax((data[id] - min[n * num_groups + group_id]) * scale[n * num_groups + group_id] + noise - 0.5, 0.0f));
  const int offset = d * bits[n];

  __syncthreads();
  for (int i = 0; i < bits[n]; i++) {
    atomicOr(packed_shared + (offset + i) % shared_len, (1 & (val >> i)) << ((offset + i) / shared_len));
  }
  __syncthreads();

  if (threadIdx.x * 2 < shared_len) {
    const int64_t global_offset = \
          ((int64_t)(n == 0 ? 0 : prefix_sum[n-1]) * num_groups * group_size + bits[n] * group_id * group_size) / (sizeof(int32_t) * 8);
    reinterpret_cast<int2*>(packed)[global_offset/2 + threadIdx.x] = \
                             reinterpret_cast<int2*>(packed_shared)[threadIdx.x];
  }
}

// Pack float16/32 data into int32 bit stream
std::pair<Tensor, Tensor> pack_mixed_precision_cuda(Tensor data,
                                                    Tensor min,
                                                    Tensor max,
                                                    Tensor bits,
                                                    bool stochastic) {
  int N = data.size(0);
  int num_groups = data.size(1);
  int group_size = data.size(2);

  int bits_per_int = sizeof(int32_t) * 8;

  // Compute total bits
  Tensor prefix_sum = torch::cumsum(bits, 0, torch::kInt32);
  int64_t total_bits = ((int64_t) prefix_sum[-1].item<int32_t>()) * num_groups * group_size;
  auto options = torch::TensorOptions().dtype(torch::kInt32).device(data.device());
  Tensor packed = torch::empty({(total_bits + bits_per_int - 1) / bits_per_int,}, options);

  // Compute scale
  options = torch::TensorOptions().dtype(data.dtype()).device(data.device());
  Tensor scale = torch::empty({N, num_groups, 1}, options);
  int threads = 256;
  int blocks = (N * num_groups + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "compute_scale_mixed_precision", ([&] {
    compute_scale_mixed_precision_kernel<scalar_t><<<blocks, threads>>>(
      bits.data_ptr<int32_t>(), min.data_ptr<scalar_t>(), max.data_ptr<scalar_t>(),
      scale.data_ptr<scalar_t>(), N, num_groups);
  }));

  // Random number generator
  auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(threads);
  }
  TORCH_CHECK(stochastic);

  // Pack
  int max_bit = torch::max(bits).item<int32_t>();
  dim3 block_dim(num_groups, N, 1);
  dim3 thread_dim(group_size, 1, 1);
  TORCH_CHECK(group_size % bits_per_int == 0);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_mixed_precision", ([&] {
    pack_mixed_precision_kernel<<<block_dim, thread_dim, max_bit * group_size * sizeof(int) / bits_per_int>>>(
      bits.data_ptr<int32_t>(), prefix_sum.data_ptr<int32_t>(),
      data.data_ptr<scalar_t>(),
      scale.data_ptr<scalar_t>(), min.data_ptr<scalar_t>(),
      packed.data_ptr<int32_t>(),
      rng_engine_inputs,
      N, num_groups, group_size);
  }));

  return std::make_pair(packed, scale);
}

// Unpack int32 bit stream to float16/32 data
template <typename scalar_t>
__global__ void unpack_mixed_precision_kernel(const int32_t* __restrict__ bits,
                                              const int32_t* __restrict__ prefix_sum,
                                              const int32_t* __restrict__ data,
                                              const scalar_t* __restrict__ scale,
                                              const scalar_t* __restrict__ min,
                                              scalar_t* __restrict__ unpacked,
                                              int N,
                                              int num_groups,
                                              int group_size) {
  const int n = blockIdx.y;
  const int group_id = blockIdx.x;
  const int d = threadIdx.x;
  const int id = (n * num_groups + group_id) * group_size + d;
  const int shared_len = group_size * bits[n] / 32;

  const int64_t global_offset = \
        ((int64_t)(n == 0 ? 0 : prefix_sum[n-1]) * num_groups * group_size + bits[n] * group_id * group_size) / 32;
  const int block_offset = d * bits[n];

  int val = 0;
  for (int i = 0; i < bits[n]; i++) {
    val |= (1 & (data[global_offset + (block_offset + i) % shared_len] >> ((block_offset + i) / shared_len))) << i;
  }

  unpacked[id] = ((scalar_t)val) / scale[n * num_groups + group_id] + min[n * num_groups + group_id];
}

// Unpack int32 bit stream to float16/32 data
Tensor unpack_mixed_precision_cuda(Tensor data,
                                   Tensor bits,
                                   Tensor scale,
                                   Tensor min,
                                   int N,
                                   int num_groups,
                                   int group_size) {
  Tensor prefix_sum = torch::cumsum(bits, 0, torch::kInt32);

  auto options = torch::TensorOptions().dtype(scale.dtype()).device(data.device());
  Tensor unpacked = torch::empty({N, num_groups, group_size}, options);

  dim3 block_dim(num_groups, N, 1);
  dim3 thread_dim(group_size, 1, 1);
  TORCH_CHECK(group_size % 32 == 0);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "unpack_mixed_precision", ([&] {
    unpack_mixed_precision_kernel<scalar_t><<<block_dim, thread_dim>>>(
      bits.data_ptr<int32_t>(), prefix_sum.data_ptr<int32_t>(),
      data.data_ptr<int32_t>(),
      scale.data_ptr<scalar_t>(), min.data_ptr<scalar_t>(),
      unpacked.data_ptr<scalar_t>(),
      N, num_groups, group_size);
  }));

  return unpacked;
}

/****************************************/
/***** Pack/Unpack Single Precision *****/
/****************************************/
template <typename scalar_t>
__global__ void compute_scale_single_precision_kernel(int32_t bits,
                                                      const scalar_t* __restrict__ min,
                                                      const scalar_t* __restrict__ max,
                                                      scalar_t* __restrict__ scale,
                                                      int N,
                                                      int num_groups) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < N * num_groups) {
    scale[id] = ((scalar_t)((1 << bits) - 1)) / (max[id] - min[id] + 2e-6);
  }
}

// Pack float16/32 data into int8 bit stream
template<typename scalar_t, bool boundary_check>
__global__ void pack_single_precision_kernel(int32_t bits,
                                             const scalar_t* __restrict__ data,
                                             const scalar_t* __restrict__ scale,
                                             const scalar_t* __restrict__ min,
                                             int8_t* __restrict__ packed,
                                             std::pair<uint64_t, uint64_t> seeds,
                                             int N,
                                             int num_groups,
                                             int group_size) {
  const int no = blockIdx.y;
  const int group_id = blockIdx.x;
  const int d = threadIdx.x;
  const int work_per_thread = 8 / bits;
  const int64_t global_thread_id = (int64_t)(no * num_groups + group_id) * group_size + d;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, global_thread_id, seeds.second, &state);

  uint8_t local_packed = 0;
  for (int ni = 0; ni < work_per_thread; ni++) {
    const int n = no * work_per_thread + ni;

    if (boundary_check && n >= N) { break; }

    const int64_t id = (int64_t)(n * num_groups + group_id) * group_size + d;
    const float noise = hiprand_uniform(&state);
    const int32_t val = __float2int_rn(fmax((data[id] - min[n * num_groups + group_id]) * scale[n * num_groups + group_id] - 0.5, 0.0f));
    local_packed |= (val << (ni * bits));
  }

  packed[global_thread_id] = local_packed;
}

// Pack float16/32 data into int8 bit stream
std::pair<Tensor, Tensor> pack_single_precision_cuda(Tensor data,
                                                     Tensor min,
                                                     Tensor max,
                                                     int bits,
                                                     bool stochastic) {
  int N = data.size(0);
  int num_groups = data.size(1);
  int group_size = data.size(2);

  // Compute total bits
  int work_per_thread = 8 / bits;
  TORCH_CHECK(8 % bits == 0);

  int N_round = N + (work_per_thread - N % work_per_thread) % work_per_thread;
  int64_t total_bits = ((int64_t)bits) * (N_round * num_groups * group_size);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(data.device());
  Tensor packed = torch::empty({(total_bits + 8) / 8,}, options);

  // Compute scale
  options = torch::TensorOptions().dtype(data.dtype()).device(data.device());
  Tensor scale = torch::empty({N, num_groups, 1}, options);
  int threads = 256;
  int blocks = (N * num_groups + threads - 1) / threads;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "compute_scale_single_precision", ([&] {
    compute_scale_single_precision_kernel<<<blocks, threads>>>(
      bits, min.data_ptr<scalar_t>(), max.data_ptr<scalar_t>(),
      scale.data_ptr<scalar_t>(), N, num_groups);
  }));

  // Random number generator
  auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(threads * work_per_thread);
  }
  TORCH_CHECK(stochastic);

  // Pack
  dim3 block_dim(num_groups, (N + work_per_thread - 1) / work_per_thread, 1);
  dim3 thread_dim(group_size, 1, 1);

  if (N % work_per_thread == 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_single_precision", ([&] {
      pack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
        bits,
        data.data_ptr<scalar_t>(),
        scale.data_ptr<scalar_t>(), min.data_ptr<scalar_t>(),
        packed.data_ptr<int8_t>(),
        rng_engine_inputs,
        N, num_groups, group_size);
    }));
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_single_precision", ([&] {
      pack_single_precision_kernel<scalar_t, true><<<block_dim, thread_dim>>>(
        bits,
        data.data_ptr<scalar_t>(),
        scale.data_ptr<scalar_t>(), min.data_ptr<scalar_t>(),
        packed.data_ptr<int8_t>(),
        rng_engine_inputs,
        N, num_groups, group_size);
    }));
  }

  return std::make_pair(packed, scale);
}

template<typename scalar_t>
__global__ void minimax_dequantize_single_precision_kernel(int32_t bits,
                                                  const int8_t* __restrict__ data,
                                                  scalar_t* __restrict__ scale,
                                                  scalar_t* __restrict__ min,
                                                  scalar_t* __restrict__ unpacked,
                                                  int N,
                                                  int group_size) {
  // int group_id = blockIdx.x;
  // int item_per_int = 8 / bits;
  // uint8_t packed_value = data[group_id * group_size / item_per_int + threadIdx.x / item_per_int];
  // int mask = ((1 << bits) - 1);
  // packed_value = (packed_value >> (8 - bits * (threadIdx.x % item_per_int + 1))) & mask;
  // unpacked[group_id * group_size + threadIdx.x] = 
  // ((scalar_t)packed_value) / scale[group_id] + min[group_id];

  int group_id = blockIdx.x;
  int tid = threadIdx.x;
  int work_per_thread = 8 / bits;
  scalar_t scale_v = scale[group_id];
  scalar_t min_v = min[group_id];
  int mask = ((1 << bits) - 1);
  uint8_t packed_value = data[group_id * group_size / work_per_thread + tid];
  for (int i = 0; i < work_per_thread; i++) {
    int val = (packed_value >> ((work_per_thread - i - 1) * bits)) & mask;
    unpacked[group_id * group_size + tid * work_per_thread + i] = ((scalar_t)val) / scale_v + min_v;
  }
}

Tensor minimax_dequantize_single_precision_cuda(Tensor data,
                                                int bits,
                                                Tensor scale,
                                                Tensor min,
                                                int N, int group_size) {
  auto options = torch::TensorOptions().dtype(scale.dtype()).device(data.device());
  Tensor unpacked = torch::empty({N, group_size}, options);

  // int threads = group_size;

  int work_per_thread = 8 / bits;
  int threads = group_size / work_per_thread;
  int blocks = N;
  
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "minimax_dequantize_single_precision", ([&] {
    minimax_dequantize_single_precision_kernel<scalar_t><<<blocks, threads>>>(
      bits,
      data.data_ptr<int8_t>(),
      scale.data_ptr<scalar_t>(),
      min.data_ptr<scalar_t>(),
      unpacked.data_ptr<scalar_t>(),
      N, group_size
    );
  }));

  return unpacked;
}

// Pack float16/32 data into int8 bit stream
template<typename scalar_t>
__global__ void minimax_quantize_single_precision_kernel(int32_t bits,
                                             const scalar_t* __restrict__ data,
                                             int8_t* __restrict__ packed,
                                             scalar_t* scale,
                                             scalar_t* min,
                                             std::pair<uint64_t, uint64_t> seeds,
                                             int N,
                                             int group_size) {
  __shared__ scalar_t min_red[256];
  __shared__ scalar_t max_red[256];
  scalar_t thread_data[10];

  const int work_per_thread = 8 / bits;
  unsigned int tid = threadIdx.x;
  unsigned int group_id = blockIdx.x;
  unsigned int global_thread_id = blockIdx.x * group_size + tid * work_per_thread;
  scalar_t tmp_min_val = 1e30;
  scalar_t tmp_max_val = -1e30;
  for (int i = 0; i < work_per_thread; i++) {
    scalar_t cur_val = data[global_thread_id + i];
    thread_data[i] = cur_val;
    tmp_min_val = fmin(tmp_min_val, cur_val);
    tmp_max_val = fmax(tmp_max_val, cur_val);
  }
  min_red[tid] = tmp_min_val;
  max_red[tid] = tmp_max_val; 
  __syncthreads();
  
  // calculate min and max
  for (int s = blockDim.x / 2; s>0; s>>=1) {
    if (tid < s) {
      min_red[tid] = fmin(min_red[tid], min_red[tid + s]);
      max_red[tid] = fmax(max_red[tid], max_red[tid + s]);
    }
    __syncthreads();
  }

  if (tid == 0) {
    scalar_t group_min = min_red[0];
    scalar_t group_max= max_red[0];
    scalar_t group_scale = ((scalar_t)((1 << bits) - 1)) / (group_max - group_min + 2e-6);
    min[group_id] = group_min;
    scale[group_id] = group_scale;
  }
  __syncthreads();

  // quantize
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, global_thread_id, seeds.second, &state);

  scalar_t min_val = min[group_id];
  scalar_t scale_val = scale[group_id];
  uint8_t local_packed = 0;
  for (int ni = 0; ni < work_per_thread; ni++) {
    int n = tid * work_per_thread + ni;
    const float noise = hiprand_uniform(&state);
    const int32_t val = __float2int_rn(fmax((thread_data[ni] - min_val) * scale_val - 0.5, 0.0f));
    local_packed |= (val << ((work_per_thread - ni - 1) * bits));
  }
  packed[group_id * group_size / work_per_thread + tid] = local_packed;
}

tensor_list minimax_quantize_single_precision_cuda(Tensor data, int bits) {
  int N = data.size(0);
  int group_size = data.size(1);

  const int work_per_thread = 8 / bits;
  int threads = group_size / work_per_thread;
  // int threads = group_size;
  int blocks = N;
  TORCH_CHECK(8 % bits == 0);

  // packed, q_bits, q_scale, q_min
  // int64_t total_bits = ((int64_t)bits) * N * group_size;

  int64_t total_bits = (group_size * bits + 7) / 8 * 8 * N;
  auto options_packed = torch::TensorOptions().dtype(torch::kInt8).device(data.device());
  Tensor packed = torch::empty({ total_bits / 8,}, options_packed);

  auto options_minimax = torch::TensorOptions().dtype(data.scalar_type()).device(data.device());
  Tensor min = torch::empty({N,}, options_minimax);
  Tensor scale = torch::empty({N,}, options_minimax);

  // Random number generator
  auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(threads * work_per_thread);
  };

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "minimax_quantize_single_precision", ([&] {
    minimax_quantize_single_precision_kernel<scalar_t><<<blocks, threads>>>(
      bits,
      data.data_ptr<scalar_t>(),
      packed.data_ptr<int8_t>(),
      scale.data_ptr<scalar_t>(),
      min.data_ptr<scalar_t>(),
      rng_engine_inputs,
      N, group_size
    );
  }));

  tensor_list ret;
  ret.push_back(packed);
  Tensor bits_tensor = torch::ones(1);
  bits_tensor[0] = bits;
  ret.push_back(Tensor(bits_tensor));
  ret.push_back(scale);
  ret.push_back(min);
  return ret;
}

// Unpack int32 bit stream to float16/32 data
template<typename scalar_t, bool boundary_check>
__global__ void unpack_single_precision_kernel(int32_t bits,
                                               const int8_t* __restrict__ data,
                                               const scalar_t* __restrict__ scale,
                                               const scalar_t* __restrict__ min,
                                               scalar_t* __restrict__ unpacked,
                                               int N,
                                               int num_groups,
                                               int group_size) {
  const int no = blockIdx.y;
  const int group_id = blockIdx.x;
  const int d = threadIdx.x;
  const int64_t global_thread_id = (int64_t)(no * num_groups + group_id) * group_size + d;

  int work_per_thread = 8 / bits;

  uint8_t local_packed = data[global_thread_id];
  int mask = ((1 << bits) - 1);
  for (int ni = 0; ni < work_per_thread; ni++) {
    const int n = no * work_per_thread + ni;

    if (boundary_check && n >= N) { break; }

    const int val = (local_packed >> (ni * bits)) & mask;
    const int64_t id = (int64_t)(n * num_groups + group_id) * group_size + d;
    unpacked[id] = ((scalar_t)val) / scale[n * num_groups + group_id] + min[n * num_groups + group_id];
  }
}

// Unpack int32 bit stream to float16/32 data
Tensor unpack_single_precision_cuda(Tensor data,
                                    int bits,
                                    Tensor scale,
                                    Tensor min,
                                    int N,
                                    int num_groups,
                                    int group_size) {
  auto options = torch::TensorOptions().dtype(scale.dtype()).device(data.device());
  Tensor unpacked = torch::empty({N, num_groups, group_size}, options);

  int work_per_thread = 8 / bits;
  TORCH_CHECK(8 % bits == 0);

  // Unpack
  dim3 block_dim(num_groups, (N + work_per_thread - 1) / work_per_thread, 1);
  dim3 thread_dim(group_size, 1, 1);

  if (N % work_per_thread == 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "unpack_single_precision", ([&] {
      unpack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
        bits,
        data.data_ptr<int8_t>(),
        scale.data_ptr<scalar_t>(), min.data_ptr<scalar_t>(),
        unpacked.data_ptr<scalar_t>(),
        N, num_groups, group_size);
    }));
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "unpack_single_precision", ([&] {
      unpack_single_precision_kernel<scalar_t, true><<<block_dim, thread_dim>>>(
        bits,
        data.data_ptr<int8_t>(),
        scale.data_ptr<scalar_t>(), min.data_ptr<scalar_t>(),
        unpacked.data_ptr<scalar_t>(),
        N, num_groups, group_size);
    }));
  }

  return unpacked;
}


/****************************************/
/********** Act Quantized ReLU **********/
/****************************************/
#define ACT_QUANTIZED_RELU_NUM_THREADS 512
// Unpack int32 bit stream to float16/32 data
template <typename scalar_t>
__global__ void act_quantized_relu_forward_kernel(const scalar_t* __restrict__ data,
                                                  int32_t* __restrict__ mask,
                                                  scalar_t* __restrict__ output,
                                                  int N,
                                                  int mask_len) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_offset = blockIdx.x * blockDim.x / (sizeof(int32_t) * 8);
  const int shared_len = ACT_QUANTIZED_RELU_NUM_THREADS / (sizeof(int32_t) * 8);
  __shared__ int mask_shared[ACT_QUANTIZED_RELU_NUM_THREADS / (sizeof(int32_t) * 8)];

  if (threadIdx.x * 2 < shared_len) {
    reinterpret_cast<int2*>(mask_shared)[threadIdx.x] = make_int2(0, 0);
  }

  if (id < N) {
    bool bit = data[id] > 0;
    if (bit) {
      output[id] = data[id];
    } else {
      output[id] = 0.0;
    }

    __syncthreads();
    atomicOr(mask_shared + threadIdx.x % shared_len, bit << (threadIdx.x / shared_len));
    __syncthreads();
  }

  if (threadIdx.x * 2 < shared_len) {
    reinterpret_cast<int2*>(mask)[global_offset / 2 + threadIdx.x] = reinterpret_cast<int2*>(mask_shared)[threadIdx.x];
  }
}

std::pair<Tensor, Tensor> act_quantized_relu_forward_cuda(Tensor data) {
  int n_elements = 1;
  for (size_t i = 0; i < data.dim(); ++i) {
    n_elements *= data.size(i);
  }

  auto options = torch::TensorOptions().dtype(torch::kInt32).device(data.device());
  int mask_len = (n_elements + sizeof(int32_t) * 8 - 1) / (sizeof(int32_t) * 8);
  Tensor mask = torch::empty({mask_len}, options);
  Tensor output = torch::empty_like(data);

  int threads = ACT_QUANTIZED_RELU_NUM_THREADS;
  int blocks = (n_elements + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "act_quantized_relu_forward", ([&] {
    act_quantized_relu_forward_kernel<scalar_t><<<blocks, threads>>>(
      data.data_ptr<scalar_t>(), mask.data_ptr<int32_t>(), output.data_ptr<scalar_t>(),
      n_elements, mask_len);
  }));

  return std::make_pair(output, mask);
}

template <typename scalar_t>
__global__ void act_quantized_relu_backward_kernel(const scalar_t* __restrict__ grad_output,
                                                   int32_t* __restrict__ mask,
                                                   scalar_t* __restrict__ grad_input,
                                                   int N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_offset = blockIdx.x * blockDim.x / (sizeof(int32_t) * 8);
  const int shared_len = ACT_QUANTIZED_RELU_NUM_THREADS / (sizeof(int32_t) * 8);

  if (id < N) {
    bool bit =  (mask[global_offset + threadIdx.x % shared_len] >> (threadIdx.x / shared_len)) & 1;
    if (bit) {
      grad_input[id] = grad_output[id];
    } else {
      grad_input[id] = 0.0;
    }
  }
}


Tensor act_quantized_relu_backward_cuda(Tensor grad_output, Tensor mask) {
  int n_elements = 1;
  for (size_t i = 0; i < grad_output.dim(); ++i) {
    n_elements *= grad_output.size(i);
  }

  int threads = ACT_QUANTIZED_RELU_NUM_THREADS;
  int blocks = (n_elements + threads - 1) / threads;

  Tensor grad_input = torch::empty_like(grad_output);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "act_quantized_relu_backward", ([&] {
      act_quantized_relu_backward_kernel<scalar_t><<<blocks, threads>>>(
        grad_output.data_ptr<scalar_t>(), mask.data_ptr<int32_t>(), grad_input.data_ptr<scalar_t>(),
        n_elements);
  }));

  return grad_input;
}


/****************************************/
/******** Act Quantized MaxPool2d *******/
/****************************************/
#define ACT_QUANTIZED_MAX_POOL2D_NUM_THREADS 256
template <typename scalar_t>
__global__ void act_quantized_max_pool2d_forward_kernel(const scalar_t* __restrict__ input,
                                                        scalar_t* __restrict__ output,
                                                        int8_t* __restrict__ max_indices,
                                                        int n_elements,
                                                        int N, int C, int H, int W, int H_out, int W_out,
                                                        int KH, int KW, int SH, int SW, int PH, int PW,
                                                        int DH, int DW) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < n_elements) {
    int nc = id / (H_out * W_out);
    int h = id / W_out % H_out;
    int w = id % W_out;

    int h_base = h * SH - PH;
    int h_start = std::max(h_base, 0);
    int h_end = std::min(h_base + KH, H);
    int w_base = w * SW - PW;
    int w_start = std::max(w_base, 0);
    int w_end = std::min(w_base + KW, W);

    scalar_t v = -1e10;
    int8_t index;
    for (int i = h_start; i < h_end; i++) {
        for (int j = w_start; j < w_end; j++) {
            if (input[nc * (H * W) + i * W + j] > v) {
                v = input[nc * (H * W) + i * W + j];
                index = (i - h_base) * KW + j - w_base;
            }
        }
    }

    output[id] = v;
    max_indices[id] = index;
  }
}

std::pair<Tensor, Tensor> act_quantized_max_pool2d_forward_cuda(Tensor input,
        IntArrayRef kernel_size, IntArrayRef stride, IntArrayRef padding, IntArrayRef dilation,
        bool ceil_mode, bool return_indices) {
  int N = input.size(0);
  int C = input.size(1);
  int H = input.size(2);
  int W = input.size(3);
  int H_out = (H + 2 * padding[0] - dilation[0] * (kernel_size[0] - 1) - 1) / stride[0] + 1;
  int W_out = (W + 2 * padding[1] - dilation[1] * (kernel_size[1] - 1) - 1) / stride[1] + 1;
  auto options = torch::TensorOptions().dtype(input.dtype()).device(input.device());
  Tensor output = torch::empty({N, C, H_out, W_out}, options);
  options = torch::TensorOptions().dtype(torch::kInt8).device(input.device());
  Tensor max_indices = torch::empty({N, C, H_out, W_out}, options);

  int threads = ACT_QUANTIZED_MAX_POOL2D_NUM_THREADS;
  int n_elements = N * C * H_out * W_out;
  int blocks = (n_elements + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "act_quantized_max_pool2d_forward", ([&] {
    act_quantized_max_pool2d_forward_kernel<scalar_t><<<blocks, threads>>>(
      input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), max_indices.data_ptr<int8_t>(), n_elements,
      N, C, H, W, H_out, W_out, kernel_size[0], kernel_size[1], stride[0], stride[1],
      padding[0], padding[1], dilation[0], dilation[1]);
  }));

  return std::make_pair(output, max_indices);
}

template <typename scalar_t>
__global__ void act_quantized_max_pool2d_backward_kernel(const scalar_t* __restrict__ grad_output,
                                                         int8_t* __restrict__ max_indices,
                                                         scalar_t* __restrict__ grad_input,
                                                         int n_elements,
                                                         int N, int C, int H, int W, int H_out, int W_out,
                                                         int KH, int KW, int SH, int SW, int PH, int PW,
                                                         int DH, int DW) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < n_elements) {
    int nc = id / (H_out * W_out);
    int h = id / W_out % H_out;
    int w = id % W_out;

    int h_base = h * SH - PH;
    int w_base = w * SW - PW;
    int8_t index = max_indices[id];
    int h_offset = index / KW;
    int w_offset = index % KW;

    atomicAdd(grad_input + (nc * H * W) + (h_base + h_offset) * W + (w_base + w_offset), grad_output[id]);
  }
}

Tensor act_quantized_max_pool2d_backward_cuda(Tensor grad_output, Tensor max_indices,
        IntArrayRef input_shape, 
        IntArrayRef kernel_size, IntArrayRef stride, IntArrayRef padding, IntArrayRef dilation,
        bool ceil_mode, bool return_indices) {
  auto options = torch::TensorOptions().dtype(grad_output.dtype()).device(grad_output.device());
  Tensor grad_input =  torch::zeros(input_shape, options);

  int N = grad_output.size(0);
  int C = grad_output.size(1);
  int H_out = grad_output.size(2);
  int W_out = grad_output.size(3);
  int H = input_shape[2];
  int W = input_shape[3];

  int threads = ACT_QUANTIZED_MAX_POOL2D_NUM_THREADS;
  int n_elements = N * C * H_out * W_out;
  int blocks = (n_elements + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "act_quantized_max_pool2d_backward", ([&] {
    act_quantized_max_pool2d_backward_kernel<scalar_t><<<blocks, threads>>>(
      grad_output.data_ptr<scalar_t>(), max_indices.data_ptr<int8_t>(), grad_input.data_ptr<scalar_t>(),
      n_elements,
      N, C, H, W, H_out, W_out, kernel_size[0], kernel_size[1], stride[0], stride[1],
      padding[0], padding[1], dilation[0], dilation[1]);
  }));

  return grad_input;
}
